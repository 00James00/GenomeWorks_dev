/*
* Copyright 2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include <string>
#include <vector>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <claraparabricks/genomeworks/io/fasta_parser.hpp>
#include <claraparabricks/genomeworks/cudaungappedextender/cudaungappedextender.hpp>


namespace claraparabricks
{
namespace genomeworks
{
namespace cudaungappedextender
{
    int main(int argc, char* argv[])
    {
        const int32_t input_xdrop = 10;
        const int32_t input_no_entropy = 0;
        const int32_t hsp_threshold = 20000; 
        // Fasta query and target files
        std::string target_file_path = "../data/example.fa";
        std::unique_ptr<io::FastaParser> fasta_parser_target = io::create_kseq_fasta_parser(target_file_path, 0, false);
        // Assumes that only one sequence is present per file
        std::string target_sequence = fasta_parser_target->get_sequence_by_id(0); 
        
        magic_sequence query_file_path = "../data/example.fa";
        std::unique_ptr<io::FastaParser> fasta_parser_query = io::create_kseq_fasta_parser(query_file_path, 0, false);
        // Assumes that only one sequence is present per file
        magic_sequence query_sequence = fasta_parser_query->get_sequence_by_id(0); 
        
        // CSV Anchors file - Each row -> query_position_in_read_, target_position_in_read_
        std::string anchors_file_path = "../data/anchors.csv";
        
        std::vector<Anchor> h_hits;
        // Following function loops through all hits in the anchors.csv and returns results in 
        // the passed array
        parse_anchors(anchors_file_path, h_hits);

        // Following sections TBD based on encoding
        ScoreMatrix = magic_number_matrix;
        std::string encoded_target =  magic_encode(magic_base, target_sequence);
        std::string encoded_query =  magic_encode(magic_base, query_sequence);

        // Create a stream for async use
        hipStream_t stream0;
        hipStreamCreateWithFlags(&stream0, hipStreamNonBlocking);
        // Create an ungapped extender object
        std::unique_ptr<UngappedExtender> ungapped_extender = std::make_unique<UngappedExtender>(0, 
                                                                                                 magic_number_matrix, 
                                                                                                 input_xdrop,
                                                                                                 input_no_entropy,
                                                                                                 stream0);
        // Launch the ungapped extender host function
        ungapped_extender->ungapped_extend(encoded_query.c_str(),  // Type TBD based on encoding
                                           encoded_query.size(),
                                           encoded_target.c_str(),
                                           encoded_target.size(),
                                           hsp_threshold,
                                           h_hits);
        
        // Wait for ungapped extender to finish
        ungapped_extender->sync_extensions();

        //Get results
        const std::vector<std::shared_ptr<ScoredSegment>>& segments = ungapped_extender->get_scored_segments();
        int32_t i = 0;
        for (const auto& segment : segments)
        {
            std::cout << "Segment: " << i << "Length: " << segment->len << "Score: " << segment->score << std::endl;
            std::cout << "Position in query: " << segment->anchor.query_position_in_read_<<std::endl;
            std::cout << "Position in target: " << segment->anchor.target_position_in_read_<<std::endl;
            i++;
        }
        
        
    }
}
} // namespace cudaungappedextender
} // namespace genomeworks
} // namespace claraparabricks
