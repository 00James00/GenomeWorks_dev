#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "matcher_gpu.cuh"

#include <cassert>
#include <numeric>

#include <thrust/scan.h>
#include <thrust/transform_scan.h>
#include <thrust/execution_policy.h>

namespace claragenomics
{

namespace cudamapper
{

MatcherGPU::MatcherGPU(const Index& query_index,
                       const Index& target_index)
{

    CGA_NVTX_RANGE(profile, "matcherGPU");
    if (query_index.unique_representations().size() == 0 || target_index.unique_representations().size() == 0)
        return;

    // We need to compute a set of anchors between the query and the target.
    // An anchor is a combination of a query (read_id, position) and
    // target {read_id, position} with the same representation.
    // The set of anchors of a matching query and target representation
    // is the all-to-all combination of the corresponding set of {(read_id, position)}
    // of the query with the set of {(read_id, position)} of the target.
    //
    // We compute the anchors for each unique representation of the query index.
    // The array index of the following data structures will correspond to the array index of the
    // unique representation in the query index.

    thrust::device_vector<std::int64_t> found_target_indices_d(query_index.unique_representations().size());
    thrust::device_vector<std::int64_t> anchor_starting_indices_d(query_index.unique_representations().size());

    // First we search for each unique representation of the query index, the array index
    // of the same representation in the array of unique representations of target index
    // (or -1 if representation is not found).
    details::matcher_gpu::find_query_target_matches(found_target_indices_d, query_index.unique_representations(), target_index.unique_representations());

    // For each unique representation of the query index compute the number of corrsponding anchors
    // and store the resulting starting index in an anchors array if all anchors are stored in a flat array.
    // The last element will be the total number of anchors.
    details::matcher_gpu::compute_anchor_starting_indices(anchor_starting_indices_d, query_index.first_occurrence_of_representations(), found_target_indices_d, target_index.first_occurrence_of_representations());

    const int64_t n_anchors = anchor_starting_indices_d.back(); // D->H transfer

    anchors_d_.resize(n_anchors);

    // Generate the anchors
    // by computing the all-to-all combinations of the matching representations in query and target
    details::matcher_gpu::generate_anchors(anchors_d_,
                                           anchor_starting_indices_d,
                                           query_index.first_occurrence_of_representations(),
                                           found_target_indices_d,
                                           target_index.first_occurrence_of_representations(),
                                           query_index.read_ids(),
                                           query_index.positions_in_reads(),
                                           target_index.read_ids(),
                                           target_index.positions_in_reads(),
                                           query_index.smallest_read_id(),
                                           target_index.smallest_read_id(),
                                           query_index.number_of_reads(),
                                           target_index.number_of_reads(),
                                           query_index.number_of_basepairs_in_longest_read(),
                                           target_index.number_of_basepairs_in_longest_read());
}

thrust::device_vector<Anchor>& MatcherGPU::anchors()
{
    return anchors_d_;
}

namespace details
{

namespace matcher_gpu
{

void find_query_target_matches(
    thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<representation_t>& query_representations_d,
    const thrust::device_vector<representation_t>& target_representations_d)
{
    assert(found_target_indices_d.size() == query_representations_d.size());

    const int32_t n_threads = 256;
    const int32_t n_blocks  = ceiling_divide<int64_t>(query_representations_d.size(), n_threads);

    find_query_target_matches_kernel<<<n_blocks, n_threads>>>(found_target_indices_d.data().get(), query_representations_d.data().get(), get_size(query_representations_d), target_representations_d.data().get(), get_size(target_representations_d));
}

void compute_anchor_starting_indices(
    thrust::device_vector<std::int64_t>& anchor_starting_indices_d,
    const thrust::device_vector<std::uint32_t>& query_starting_index_of_each_representation_d,
    const thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<std::uint32_t>& target_starting_index_of_each_representation_d)
{
    assert(query_starting_index_of_each_representation_d.size() == found_target_indices_d.size() + 1);
    assert(anchor_starting_indices_d.size() == found_target_indices_d.size());

    const std::uint32_t* const query_starting_indices  = query_starting_index_of_each_representation_d.data().get();
    const std::uint32_t* const target_starting_indices = target_starting_index_of_each_representation_d.data().get();
    const std::int64_t* const found_target_indices     = found_target_indices_d.data().get();

    thrust::transform_inclusive_scan(
        thrust::make_counting_iterator(std::int64_t(0)),
        thrust::make_counting_iterator(get_size(anchor_starting_indices_d)),
        anchor_starting_indices_d.begin(),
        [query_starting_indices, target_starting_indices, found_target_indices] __device__(std::uint32_t query_index) -> std::int64_t {
            std::int32_t n_queries_with_representation = query_starting_indices[query_index + 1] - query_starting_indices[query_index];
            std::int64_t target_index                  = found_target_indices[query_index];
            std::int32_t n_targets_with_representation = 0;
            if (target_index >= 0)
                n_targets_with_representation = target_starting_indices[target_index + 1] - target_starting_indices[target_index];
            return n_queries_with_representation * n_targets_with_representation;
        },
        thrust::plus<std::int64_t>());
}

void generate_anchors(
    thrust::device_vector<Anchor>& anchors,
    const thrust::device_vector<std::int64_t>& anchor_starting_indices_d,
    const thrust::device_vector<std::uint32_t>& query_starting_index_of_each_representation_d,
    const thrust::device_vector<std::int64_t>& found_target_indices_d,
    const thrust::device_vector<std::uint32_t>& target_starting_index_of_each_representation_d,
    const thrust::device_vector<read_id_t>& query_read_ids,
    const thrust::device_vector<position_in_read_t>& query_positions_in_read,
    const thrust::device_vector<read_id_t>& target_read_ids,
    const thrust::device_vector<position_in_read_t>& target_positions_in_read,
    const read_id_t smallest_query_read_id,
    const read_id_t smallest_target_read_id,
    const read_id_t number_of_query_reads,
    const read_id_t number_of_target_reads,
    const position_in_read_t max_basepairs_in_query_reads,
    const position_in_read_t max_basepairs_in_target_reads)
{
    assert(anchor_starting_indices_d.size() + 1 == query_starting_index_of_each_representation_d.size());
    assert(found_target_indices_d.size() + 1 == query_starting_index_of_each_representation_d.size());
    assert(query_read_ids.size() == query_positions_in_read.size());
    assert(target_read_ids.size() == target_positions_in_read.size());

    std::uint64_t max_reads_compound_key     = number_of_query_reads * static_cast<std::uint64_t>(number_of_target_reads) + number_of_target_reads;
    std::uint64_t max_positions_compound_key = max_basepairs_in_query_reads * static_cast<std::uint64_t>(max_basepairs_in_target_reads) + max_basepairs_in_target_reads;

    // TODO: This solution with four separate calls depending on max key sizes ir rather messy.
    //       Look for a solution similar to std::conditional, but which can be done at runtime.
    //       Alternatively pack repreted calls into a tempatized function.

    bool reads_compound_key_32_bit     = max_reads_compound_key <= std::numeric_limits<std::uint32_t>::max();
    bool positions_compound_key_32_bit = max_positions_compound_key <= std::numeric_limits<std::uint32_t>::max();

    if (reads_compound_key_32_bit)
    {
        using ReadsKeyT = std::uint32_t;
        if (positions_compound_key_32_bit)
        {
            using PositionsKeyT = std::uint32_t;

            thrust::device_vector<ReadsKeyT> compound_key_read_ids;
            thrust::device_vector<PositionsKeyT> compound_key_positions_in_reads;

            details::matcher_gpu::generate_partially_sorted_anchors<ReadsKeyT, PositionsKeyT>(anchors,
                                                                                              compound_key_read_ids,
                                                                                              compound_key_positions_in_reads,
                                                                                              anchor_starting_indices_d,
                                                                                              query_starting_index_of_each_representation_d,
                                                                                              found_target_indices_d,
                                                                                              target_starting_index_of_each_representation_d,
                                                                                              query_read_ids,
                                                                                              query_positions_in_read,
                                                                                              target_read_ids,
                                                                                              target_positions_in_read,
                                                                                              smallest_query_read_id,
                                                                                              smallest_target_read_id,
                                                                                              number_of_target_reads,
                                                                                              max_basepairs_in_target_reads);

            {
                CGA_NVTX_RANGE(profile, "matcherGPU::sort_anchors");
                // sort anchors by query_read_id -> target_read_id -> query_position_in_read -> target_position_in_read
                cudautils::sort_by_two_keys(compound_key_read_ids,
                                            compound_key_positions_in_reads,
                                            anchors,
                                            static_cast<ReadsKeyT>(max_reads_compound_key),
                                            static_cast<PositionsKeyT>(max_positions_compound_key));
            }
        }
        else
        {
            using PositionsKeyT = std::uint64_t;

            thrust::device_vector<ReadsKeyT> compound_key_read_ids;
            thrust::device_vector<PositionsKeyT> compound_key_positions_in_reads;

            details::matcher_gpu::generate_partially_sorted_anchors<ReadsKeyT, PositionsKeyT>(anchors,
                                                                                              compound_key_read_ids,
                                                                                              compound_key_positions_in_reads,
                                                                                              anchor_starting_indices_d,
                                                                                              query_starting_index_of_each_representation_d,
                                                                                              found_target_indices_d,
                                                                                              target_starting_index_of_each_representation_d,
                                                                                              query_read_ids,
                                                                                              query_positions_in_read,
                                                                                              target_read_ids,
                                                                                              target_positions_in_read,
                                                                                              smallest_query_read_id,
                                                                                              smallest_target_read_id,
                                                                                              number_of_target_reads,
                                                                                              max_basepairs_in_target_reads);

            {
                CGA_NVTX_RANGE(profile, "matcherGPU::sort_anchors");
                // sort anchors by query_read_id -> target_read_id -> query_position_in_read -> target_position_in_read
                cudautils::sort_by_two_keys(compound_key_read_ids,
                                            compound_key_positions_in_reads,
                                            anchors,
                                            static_cast<ReadsKeyT>(max_reads_compound_key),
                                            static_cast<PositionsKeyT>(max_positions_compound_key));
            }
        }
    }
    else
    {
        using ReadsKeyT = std::uint64_t;
        if (positions_compound_key_32_bit)
        {
            using PositionsKeyT = std::uint32_t;
            thrust::device_vector<ReadsKeyT> compound_key_read_ids;
            thrust::device_vector<PositionsKeyT> compound_key_positions_in_reads;

            details::matcher_gpu::generate_partially_sorted_anchors<ReadsKeyT, PositionsKeyT>(anchors,
                                                                                              compound_key_read_ids,
                                                                                              compound_key_positions_in_reads,
                                                                                              anchor_starting_indices_d,
                                                                                              query_starting_index_of_each_representation_d,
                                                                                              found_target_indices_d,
                                                                                              target_starting_index_of_each_representation_d,
                                                                                              query_read_ids,
                                                                                              query_positions_in_read,
                                                                                              target_read_ids,
                                                                                              target_positions_in_read,
                                                                                              smallest_query_read_id,
                                                                                              smallest_target_read_id,
                                                                                              number_of_target_reads,
                                                                                              max_basepairs_in_target_reads);

            {
                CGA_NVTX_RANGE(profile, "matcherGPU::sort_anchors");
                // sort anchors by query_read_id -> target_read_id -> query_position_in_read -> target_position_in_read
                cudautils::sort_by_two_keys(compound_key_read_ids,
                                            compound_key_positions_in_reads,
                                            anchors,
                                            static_cast<ReadsKeyT>(max_reads_compound_key),
                                            static_cast<PositionsKeyT>(max_positions_compound_key));
            }
        }
        else
        {
            using PositionsKeyT = std::uint64_t;

            thrust::device_vector<ReadsKeyT> compound_key_read_ids;
            thrust::device_vector<PositionsKeyT> compound_key_positions_in_reads;

            details::matcher_gpu::generate_partially_sorted_anchors<ReadsKeyT, PositionsKeyT>(anchors,
                                                                                              compound_key_read_ids,
                                                                                              compound_key_positions_in_reads,
                                                                                              anchor_starting_indices_d,
                                                                                              query_starting_index_of_each_representation_d,
                                                                                              found_target_indices_d,
                                                                                              target_starting_index_of_each_representation_d,
                                                                                              query_read_ids,
                                                                                              query_positions_in_read,
                                                                                              target_read_ids,
                                                                                              target_positions_in_read,
                                                                                              smallest_query_read_id,
                                                                                              smallest_target_read_id,
                                                                                              number_of_target_reads,
                                                                                              max_basepairs_in_target_reads);

            {
                CGA_NVTX_RANGE(profile, "matcherGPU::sort_anchors");
                // sort anchors by query_read_id -> target_read_id -> query_position_in_read -> target_position_in_read
                cudautils::sort_by_two_keys(compound_key_read_ids,
                                            compound_key_positions_in_reads,
                                            anchors,
                                            static_cast<ReadsKeyT>(max_reads_compound_key),
                                            static_cast<PositionsKeyT>(max_positions_compound_key));
            }
        }
    }
}

__global__ void find_query_target_matches_kernel(
    int64_t* const found_target_indices,
    const representation_t* const query_representations_d,
    const int64_t n_query_representations,
    const representation_t* const target_representations_d,
    const int64_t n_target_representations)
{
    const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n_query_representations)
        return;

    const representation_t query = query_representations_d[i];
    int64_t found_target_index   = -1;
    const representation_t* lb   = lower_bound(target_representations_d, target_representations_d + n_target_representations, query);
    if (*lb == query)
        found_target_index = lb - target_representations_d;

    found_target_indices[i] = found_target_index;
}

} // namespace matcher_gpu

} // namespace details
} // namespace cudamapper

} // namespace claragenomics
