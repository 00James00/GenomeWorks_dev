#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "matcher_gpu.cuh"

namespace claragenomics
{

namespace cudamapper
{

MatcherGPU::MatcherGPU(const Index& query_index,
                       const Index& target_index)
{
}

std::vector<Anchor>& MatcherGPU::anchors()
{
    return anchors_h_;
}

namespace details
{

namespace matcher_gpu
{

__global__ void create_new_value_mask(const representation_t* const representations_d,
                                      const std::size_t number_of_elements,
                                      std::uint8_t* const new_value_mask_d)
{
    std::uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= number_of_elements)
        return;

    if (index == 0)
    {
        new_value_mask_d[0] = 1;
    }
    else
    {
        if (representations_d[index] == representations_d[index - 1])
        {
            new_value_mask_d[index] = 0;
        }
        else
            new_value_mask_d[index] = 1;
    }
}

} // namespace matcher_gpu

} // namespace details
} // namespace cudamapper

} // namespace claragenomics
