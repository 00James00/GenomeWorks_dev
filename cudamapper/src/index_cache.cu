/*
* Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include "index_cache.cuh"

#include "index_host_copy.cuh"

#include <unordered_set>

#include <claraparabricks/genomeworks/cudamapper/index.hpp>
#include <claraparabricks/genomeworks/io/fasta_parser.hpp>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudamapper
{

IndexCacheHost::IndexCacheHost(const bool same_query_and_target,
                               genomeworks::DefaultDeviceAllocator allocator,
                               std::shared_ptr<genomeworks::io::FastaParser> query_parser,
                               std::shared_ptr<genomeworks::io::FastaParser> target_parser,
                               const std::uint64_t kmer_size,
                               const std::uint64_t window_size,
                               const bool hash_representations,
                               const double filtering_parameter,
                               const hipStream_t cuda_stream_generation,
                               const hipStream_t cuda_stream_copy)
    : same_query_and_target_(same_query_and_target)
    , allocator_(allocator)
    , query_parser_(query_parser)
    , target_parser_(target_parser)
    , kmer_size_(kmer_size)
    , window_size_(window_size)
    , hash_representations_(hash_representations)
    , filtering_parameter_(filtering_parameter)
    , cuda_stream_generation_(cuda_stream_generation)
    , cuda_stream_copy_(cuda_stream_copy)
{
}

void IndexCacheHost::start_generating_query_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                                          const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                                          const bool skip_copy_to_host)
{
    start_generating_cache_content(descriptors_of_indices_to_cache,
                                   descriptors_of_indices_to_keep_on_device,
                                   skip_copy_to_host,
                                   CacheSelector::query_cache);
}

void IndexCacheHost::finish_generating_query_cache_content()
{
    finish_generating_cache_content(CacheSelector::query_cache);
    return;
}

void IndexCacheHost::start_generating_target_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                                           const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                                           const bool skip_copy_to_host)
{
    start_generating_cache_content(descriptors_of_indices_to_cache,
                                   descriptors_of_indices_to_keep_on_device,
                                   skip_copy_to_host,
                                   CacheSelector::target_cache);
}

void IndexCacheHost::finish_generating_target_cache_content()
{
    finish_generating_cache_content(CacheSelector::target_cache);
    return;
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_query_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    return get_index_from_cache(descriptor_of_index_to_cache,
                                CacheSelector::query_cache);
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_target_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    return get_index_from_cache(descriptor_of_index_to_cache,
                                CacheSelector::target_cache);
}

void IndexCacheHost::start_generating_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                                    const std::vector<IndexDescriptor>& descriptors_of_indices_to_keep_on_device,
                                                    const bool skip_copy_to_host,
                                                    const CacheSelector which_cache)
{
    // skip_copy_to_host only makes sense if descriptors_of_indices_to_cache and descriptors_of_indices_to_keep_on_device are the same
    // otherwise some indices would be created and not saved on either host or device
    assert(!skip_copy_to_host || (descriptors_of_indices_to_cache == descriptors_of_indices_to_keep_on_device));

    cache_type_t& cache_to_edit                                                = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    const cache_type_t& cache_to_check                                         = (CacheSelector::query_cache == which_cache) ? target_cache_ : query_cache_;
    device_cache_type_t& temp_device_cache_to_edit                             = (CacheSelector::query_cache == which_cache) ? query_temp_device_cache_ : target_temp_device_cache_;
    const device_cache_type_t& temp_device_cache_to_check                      = (CacheSelector::query_cache == which_cache) ? target_temp_device_cache_ : query_temp_device_cache_;
    const genomeworks::io::FastaParser* parser                                 = (CacheSelector::query_cache == which_cache) ? query_parser_.get() : target_parser_.get();
    std::vector<std::shared_ptr<const IndexHostCopyBase>>& indices_in_progress = (CacheSelector::query_cache == which_cache) ? query_indices_in_progress : target_indices_in_progress;

    // convert descriptors_of_indices_to_keep_on_device into set for faster search
    std::unordered_set<IndexDescriptor, IndexDescriptorHash> descriptors_of_indices_to_keep_on_device_set(begin(descriptors_of_indices_to_keep_on_device),
                                                                                                          end(descriptors_of_indices_to_keep_on_device));

    cache_type_t new_cache;
    temp_device_cache_to_edit.clear(); // this should be empty by now anyway

    for (const IndexDescriptor& descriptor_of_index_to_cache : descriptors_of_indices_to_cache)
    {
        // check if this index should be kept on device in addition to copying it to host
        const bool keep_on_device = descriptors_of_indices_to_keep_on_device_set.count(descriptor_of_index_to_cache) != 0;

        std::shared_ptr<const IndexHostCopyBase> index_on_host = nullptr;
        std::shared_ptr<Index> index_on_device                 = nullptr;

        // if query and target are the same check if this index is already cached in the other cache
        if (same_query_and_target_)
        {
            auto existing_cache = cache_to_check.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_check.end())
            {
                index_on_host = existing_cache->second;
                if (keep_on_device)
                {
                    auto existing_device_cache = temp_device_cache_to_check.find(descriptor_of_index_to_cache);
                    if (existing_device_cache != temp_device_cache_to_check.end())
                    {
                        index_on_device = existing_device_cache->second;
                    }
                    else
                    {
                        index_on_device = index_on_host->copy_index_to_device(allocator_, cuda_stream_generation_);
                    }
                }
            }
        }

        // if index was not found in the other cache check current cache and generate index if needed
        if (nullptr == index_on_host)
        {
            auto existing_cache = cache_to_edit.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_edit.end())
            {
                // index already cached
                index_on_host = existing_cache->second;
                if (keep_on_device)
                {
                    index_on_device = index_on_host->copy_index_to_device(allocator_, cuda_stream_generation_);
                }
            }
            else
            {
                // create index
                index_on_device = Index::create_index(allocator_,
                                                      *parser,
                                                      descriptor_of_index_to_cache.first_read(),
                                                      descriptor_of_index_to_cache.first_read() + descriptor_of_index_to_cache.number_of_reads(),
                                                      kmer_size_,
                                                      window_size_,
                                                      hash_representations_,
                                                      filtering_parameter_,
                                                      cuda_stream_generation_);

                // copy it to host memory
                if (!skip_copy_to_host)
                {
                    // wait for index to be generated on cuda_stream_generation_ before copying it on cuda_stream_copy_
                    // TODO: do this sync using an event
                    GW_CU_CHECK_ERR(hipStreamSynchronize(cuda_stream_generation_));

                    index_on_host = IndexHostCopy::create_cache(*index_on_device,
                                                                descriptor_of_index_to_cache.first_read(),
                                                                kmer_size_,
                                                                window_size_,
                                                                cuda_stream_copy_);
                    // index is being copied from device to host which is done asynchronously
                    indices_in_progress.push_back(index_on_host);
                }
            }
        }

        // save pointer to cached index
        if (!skip_copy_to_host)
        {
            assert(nullptr != index_on_host);
            new_cache[descriptor_of_index_to_cache] = index_on_host;
        }

        if (keep_on_device)
        {
            temp_device_cache_to_edit[descriptor_of_index_to_cache] = index_on_device;
        }
    }

    std::swap(new_cache, cache_to_edit);
}

void IndexCacheHost::finish_generating_cache_content(CacheSelector which_cache)
{
    std::vector<std::shared_ptr<const IndexHostCopyBase>>& indices_in_progress = (CacheSelector::query_cache == which_cache) ? query_indices_in_progress : target_indices_in_progress;

    // wait for all indices to be generated
    GW_CU_CHECK_ERR(hipStreamSynchronize(cuda_stream_generation_));

    // wait for all indices to finish being copied to host memory
    for (auto& index_in_progress : indices_in_progress)
    {
        index_in_progress->finish_copying_to_host();
    }
    indices_in_progress.clear();
}

std::shared_ptr<Index> IndexCacheHost::get_index_from_cache(const IndexDescriptor& descriptor_of_index_to_cache,
                                                            const CacheSelector which_cache)
{
    std::shared_ptr<Index> index;

    const cache_type_t& host_cache               = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    device_cache_type_t& temp_device_index_cache = (CacheSelector::query_cache == which_cache) ? query_temp_device_cache_ : target_temp_device_cache_;

    auto temp_device_index_cache_iter = temp_device_index_cache.find(descriptor_of_index_to_cache);
    // check if index is present in device memory, copy from host if not
    if (temp_device_index_cache_iter != temp_device_index_cache.end())
    {
        index = temp_device_index_cache_iter->second;
        // indices are removed from device cache after they have been used for the first time
        temp_device_index_cache.erase(temp_device_index_cache_iter);
    }
    else
    {
        // TODO: throw custom exception if index not found
        index = host_cache.at(descriptor_of_index_to_cache)->copy_index_to_device(allocator_, cuda_stream_generation_);
    }

    return index;
}

IndexCacheDevice::IndexCacheDevice(const bool same_query_and_target,
                                   std::shared_ptr<IndexCacheHost> index_cache_host)
    : same_query_and_target_(same_query_and_target)
    , index_cache_host_(index_cache_host)
{
}

void IndexCacheDevice::generate_query_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache)
{
    generate_cache_content(descriptors_of_indices_to_cache, CacheSelector::query_cache);
}

void IndexCacheDevice::generate_target_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache)
{
    generate_cache_content(descriptors_of_indices_to_cache, CacheSelector::target_cache);
}

std::shared_ptr<Index> IndexCacheDevice::get_index_from_query_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    // TODO: throw custom exception if index not found
    return query_cache_.at(descriptor_of_index_to_cache);
}

std::shared_ptr<Index> IndexCacheDevice::get_index_from_target_cache(const IndexDescriptor& descriptor_of_index_to_cache)
{
    // TODO: throw custom exception if index not found
    return target_cache_.at(descriptor_of_index_to_cache);
}

void IndexCacheDevice::generate_cache_content(const std::vector<IndexDescriptor>& descriptors_of_indices_to_cache,
                                              const CacheSelector which_cache)
{
    cache_type_t& cache_to_edit        = (CacheSelector::query_cache == which_cache) ? query_cache_ : target_cache_;
    const cache_type_t& cache_to_check = (CacheSelector::query_cache == which_cache) ? target_cache_ : query_cache_;

    cache_type_t new_cache;

    for (const IndexDescriptor& descriptor_of_index_to_cache : descriptors_of_indices_to_cache)
    {

        std::shared_ptr<Index> index = nullptr;

        if (same_query_and_target_)
        {
            // check if the same index already exists in the other cache
            auto existing_cache = cache_to_check.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_check.end())
            {
                index = existing_cache->second;
            }
        }

        if (nullptr == index)
        {
            // check if this index is already cached in this cache
            auto existing_cache = cache_to_edit.find(descriptor_of_index_to_cache);
            if (existing_cache != cache_to_edit.end())
            {
                // index already cached
                index = existing_cache->second;
            }
            else
            {
                // index not already cached -> fetch it from index_cache_host_
                if (CacheSelector::query_cache == which_cache)
                {
                    index = index_cache_host_->get_index_from_query_cache(descriptor_of_index_to_cache);
                }
                else
                {
                    index = index_cache_host_->get_index_from_target_cache(descriptor_of_index_to_cache);
                }
            }
        }

        assert(nullptr != index);

        // save pointer to cached index
        new_cache[descriptor_of_index_to_cache] = index;
    }

    std::swap(new_cache, cache_to_edit);
}

} // namespace cudamapper

} // namespace genomeworks

} // namespace claraparabricks
