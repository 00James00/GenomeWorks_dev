#include "hip/hip_runtime.h"
/*
* Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
*
* NVIDIA CORPORATION and its licensors retain all intellectual property
* and proprietary rights in and to this software, related documentation
* and any modifications thereto.  Any use, reproduction, disclosure or
* distribution of this software and related documentation without an express
* license agreement from NVIDIA CORPORATION is strictly prohibited.
*/

#include <chrono>
#include <getopt.h>
#include <iostream>
#include <string>
#include <deque>
#include <mutex>
#include <future>
#include <thread>
#include <atomic>
#include <map>

#include "ThreadPool.h"

#include <claragenomics/logging/logging.hpp>
#include <claragenomics/io/fasta_parser.hpp>
#include <claragenomics/utils/cudautils.hpp>

#include <claragenomics/cudamapper/index.hpp>
#include <claragenomics/cudamapper/matcher.hpp>
#include <claragenomics/cudamapper/overlapper.hpp>
#include "overlapper_triggered.hpp"

static struct option options[] = {
    {"window-size", required_argument, 0, 'w'},
    {"kmer-size", required_argument, 0, 'k'},
    {"num-devices", required_argument, 0, 'd'},
    {"index-size", required_argument, 0, 'i'},
    {"target-index-size", required_argument, 0, 't'},
    {"max-cache-size", required_argument, 0, 'c'},
    {"help", no_argument, 0, 'h'},
};

void help(int32_t exit_code);

int main(int argc, char* argv[])
{
    claragenomics::logging::Init();

    uint32_t k                     = 15;
    uint32_t w                     = 15;
    std::int32_t index_size        = 10000;
    std::int32_t num_devices       = 1;
    std::int32_t target_index_size = 10000;
    std::int32_t max_cache_size    = 100;
    std::string optstring          = "t:i:k:w:h:d:c:";
    uint32_t argument;
    while ((argument = getopt_long(argc, argv, optstring.c_str(), options, nullptr)) != -1)
    {
        switch (argument)
        {
        case 'k':
            k = atoi(optarg);
            break;
        case 'w':
            w = atoi(optarg);
            break;
        case 'i':
            index_size = atoi(optarg);
            break;
        case 'd':
            num_devices = atoi(optarg);
            break;
        case 't':
            target_index_size = atoi(optarg);
            break;
        case 'c':
            max_cache_size = atoi(optarg);
            break;
        case 'h':
            help(0);
        default:
            exit(1);
        }
    }

    if (k > claragenomics::cudamapper::Index::maximum_kmer_size())
    {
        std::cerr << "kmer of size " << k << " is not allowed, maximum k = " << claragenomics::cudamapper::Index::maximum_kmer_size() << std::endl;
        exit(1);
    }

    // Check remaining argument count.
    if ((argc - optind) < 2)
    {
        std::cerr << "Invalid inputs. Please refer to the help function." << std::endl;
        help(1);
    }

    std::string query_filepath  = std::string(argv[optind++]);
    std::string target_filepath = std::string(argv[optind++]);

    bool all_to_all = false;
    if (query_filepath == target_filepath)
    {
        all_to_all        = true;
        target_index_size = index_size;
        std::cerr << "NOTE - Since query and target files are same, activating all_to_all mode. Query index size used for both files." << std::endl;
    }

    std::unique_ptr<claragenomics::io::FastaParser> query_parser = claragenomics::io::create_fasta_parser(query_filepath);
    int32_t queries                                              = query_parser->get_num_seqences();

    std::unique_ptr<claragenomics::io::FastaParser> target_parser = claragenomics::io::create_fasta_parser(target_filepath);
    int32_t targets                                               = target_parser->get_num_seqences();

    std::cerr << "Query " << query_filepath << " index " << queries << std::endl;
    std::cerr << "Target " << target_filepath << " index " << targets << std::endl;

    // Data structure for holding overlaps to be written out
    std::mutex overlaps_writer_mtx;
    std::mutex index_cache_mtx;

    struct query_target_range
    {
        std::pair<std::int32_t, int32_t> query_range;
        std::vector<std::pair<std::int32_t, int32_t>> target_ranges;
    };

    //First generate all the ranges independently, then loop over them.
    std::vector<query_target_range> query_target_ranges;

    for (std::int32_t query_start_index = 0; query_start_index < queries; query_start_index += index_size)
    {

        std::int32_t query_end_index = std::min(query_start_index + index_size, queries);

        query_target_range q;
        q.query_range = std::make_pair(query_start_index, query_end_index);

        std::int32_t target_start_index = 0;
        // If all_to_all mode, then we can optimzie by starting the target sequences from the same index as
        // query because all indices before the current query index are guaranteed to have been processed in
        // a2a mapping.
        if (all_to_all)
        {
            target_start_index = query_start_index;
        }

        for (; target_start_index < targets; target_start_index += target_index_size)
        {
            std::int32_t target_end_index = std::min(target_start_index + target_index_size,
                                                     targets);
            q.target_ranges.push_back(std::make_pair(target_start_index, target_end_index));
        }

        query_target_ranges.push_back(q);
    }

    // This is a per-device cache, if it has the index it will return it, if not it will generate it, store and return it.
    std::vector<std::map<std::pair<uint64_t, uint64_t>, std::shared_ptr<claragenomics::cudamapper::Index>>> index_cache(num_devices);

    auto get_index = [&index_cache, &index_cache_mtx, max_cache_size](claragenomics::io::FastaParser& parser,
                                                                      const claragenomics::cudamapper::read_id_t start_index,
                                                                      const claragenomics::cudamapper::read_id_t end_index,
                                                                      const std::uint64_t k,
                                                                      const std::uint64_t w,
                                                                      const int device_id,
                                                                      const bool all_to_all) {
        CGA_NVTX_RANGE(profiler, "get index");
        std::pair<uint64_t, uint64_t> key;
        key.first  = start_index;
        key.second = end_index;

        std::shared_ptr<claragenomics::cudamapper::Index> index;

        if (index_cache[device_id].count(key))
        {
            index = index_cache[device_id][key];
        }
        else
        {
            index = std::move(claragenomics::cudamapper::Index::create_index(parser, start_index, end_index, k, w));

            // If in all-to-all mode, put this query in the cache for later use.
            // Cache eviction is handled later on by the calling thread
            // using the evict_index function.
            if (index_cache[device_id].size() < max_cache_size && all_to_all)
            {
                index_cache_mtx.lock();
                index_cache[device_id][key] = index;
                index_cache_mtx.unlock();
            }
        }
        return index;
    };

    // When performing all-to-all mapping, indices are instantitated as start-end-ranges in the reads.
    // As such, once a query index has been used it will not be needed again. For example, parsing ranges
    // [0-999], [1000-1999], [2000-2999], the caching/eviction would be as follows:
    //
    // Round 1
    // Query: [0-999] - Enter cache
    // Target: [1000-1999] - Enter cache
    // Target: [1999 - 2999] - Enter cache
    // Evict [0-999]
    // Round 2
    // Query: [1000-1999] - Use cache entry (from previous use when now query was a target)
    // Etc..
    auto evict_index = [&index_cache, &index_cache_mtx](const claragenomics::cudamapper::read_id_t query_start_index,
                                                        const claragenomics::cudamapper::read_id_t query_end_index,
                                                        const int device_id) {
        std::pair<uint64_t, uint64_t> key;
        key.first  = query_start_index;
        key.second = query_end_index;

        index_cache_mtx.lock();
        index_cache[device_id].erase(key);
        index_cache_mtx.unlock();
    };

    auto compute_overlaps = [&](const query_target_range query_target_range, const int device_id) {
        std::vector<std::shared_ptr<std::future<void>>> print_pafs_futures;

        hipSetDevice(device_id);

        auto query_start_index = query_target_range.query_range.first;
        auto query_end_index   = query_target_range.query_range.second;

        std::cerr << "Procecssing query range: (" << query_start_index << " - " << query_end_index - 1 << ")" << std::endl;

        std::shared_ptr<claragenomics::cudamapper::Index> query_index(nullptr);
        std::shared_ptr<claragenomics::cudamapper::Index> target_index(nullptr);
        std::unique_ptr<claragenomics::cudamapper::Matcher> matcher(nullptr);

        {
            CGA_NVTX_RANGE(profiler, "generate_query_index");
            auto start_time = std::chrono::high_resolution_clock::now();

            query_index = get_index(*query_parser, query_start_index, query_end_index, k, w, device_id, all_to_all);
        }

        //Main loop
        for (const auto target_range : query_target_range.target_ranges)
        {

            auto target_start_index = target_range.first;
            auto target_end_index   = target_range.second;

            {
                CGA_NVTX_RANGE(profiler, "generate_target_index");
                auto start_time = std::chrono::high_resolution_clock::now();
                target_index    = get_index(*target_parser, target_start_index, target_end_index, k, w, device_id, all_to_all);
            }
            {
                CGA_NVTX_RANGE(profiler, "generate_matcher");
                auto start_time = std::chrono::high_resolution_clock::now();
                matcher         = claragenomics::cudamapper::Matcher::create_matcher(*query_index,
                                                                             *target_index);
            }
            {

                claragenomics::cudamapper::OverlapperTriggered overlapper;
                CGA_NVTX_RANGE(profiler, "generate_overlaps");
                auto start_time = std::chrono::high_resolution_clock::now();

                // Get unfiltered overlaps
                std::vector<claragenomics::cudamapper::Overlap> overlaps_to_add;
                overlapper.get_overlaps(overlaps_to_add, matcher->anchors(), *query_index, *target_index);

                std::shared_ptr<std::future<void>> write_and_filter_overlaps_future = std::make_shared<std::future<void>>(std::async(std::launch::async,
                                                                                                                                     [&overlaps_writer_mtx, overlaps_to_add](std::vector<claragenomics::cudamapper::Overlap> overlaps) {
                                                                                                                                         std::vector<claragenomics::cudamapper::Overlap> filtered_overlaps;
                                                                                                                                         claragenomics::cudamapper::Overlapper::filter_overlaps(filtered_overlaps, overlaps_to_add);
                                                                                                                                         overlaps_writer_mtx.lock();
                                                                                                                                         claragenomics::cudamapper::Overlapper::print_paf(filtered_overlaps);
                                                                                                                                         overlaps_writer_mtx.unlock();
                                                                                                                                     },
                                                                                                                                     overlaps_to_add));

                print_pafs_futures.push_back(write_and_filter_overlaps_future);
            }
        }

        //Query will no longer be needed on device, remove it from the cache
        evict_index(query_start_index, query_end_index, device_id);
        return print_pafs_futures;
    };

    // The application (File parsing, index generation, overlap generation etc) is all launched from here.
    // The main application works as follows:
    // 1. Generate a thread pool with one worker thread per device (GPU).
    // 2. For each worker, assign a query-target-range, this is list of query reads and a *list of lists of target reads* to match against.
    // 3. Each worker returns a vector of futures (since overlap writing is dispatched to an async thread on host). All futures are waited for before the main application exits.

    // 1. create thread pool to compute overlaps. One worker thread per device.
    ThreadPool overlap_pool(num_devices);

    // 2. Enqueue the query-target ranges which need to be computed, each thread returns a vector of futures for the threads it launches
    std::vector<std::future<std::vector<std::shared_ptr<std::future<void>>>>> overlap_futures;
    for (int i = 0; i < query_target_ranges.size(); i++)
    {
        // assign chunk pairs (Query, target) to device
        auto query_target_range = query_target_ranges[i];
        auto device_id          = i % num_devices;
        overlap_futures.push_back(overlap_pool.enqueue(compute_overlaps, query_target_range, device_id));
    }

    // 3. Wait for all futures.
    for (auto& overlap_future : overlap_futures)
    {
        for (auto query_target_pair_overlaps : overlap_future.get())
        {
            query_target_pair_overlaps->wait();
        }
    }

    return 0;
}

void help(int32_t exit_code = 0)
{
    std::cerr <<
        R"(Usage: cudamapper [options ...] <query_sequences> <target_sequences>
     <sequences>
        Input file in FASTA/FASTQ format (can be compressed with gzip)
        containing sequences used for all-to-all overlapping
     options:
        -k, --kmer-size
            length of kmer to use for minimizers [15] (Max=)"
              << claragenomics::cudamapper::Index::maximum_kmer_size() << ")"
              << R"(
        -w, --window-size
            length of window to use for minimizers [15])"
              << R"(
        -d, --num-devices
            number of GPUs to use [1])"
              << R"(
        -c, --max_cache_size
            number of indices to keep in GPU memory [100])"
              << R"(
        -i, --index-size
            length of batch size used for query [10000])"
              << R"(
        -t --target-index-size
            length of batch sized used for target [10000])"
              << std::endl;

    exit(exit_code);
}
