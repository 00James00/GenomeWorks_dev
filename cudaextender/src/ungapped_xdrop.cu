#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include <claraparabricks/genomeworks/utils/pinned_host_vector.hpp>
#include <claraparabricks/genomeworks/utils/cudautils.hpp>
#include "ungapped_xdrop.cuh"
#include "ungapped_xdrop_kernels.cuh"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/system/cpp/execution_policy.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/distance.h>
#include <thrust/device_vector.h>
#include <cub/device/device_select.cuh>
#include <chrono>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudaextender
{

using namespace cudautils;

UngappedXDrop::UngappedXDrop(int32_t* h_sub_mat, int32_t sub_mat_dim, int32_t xdrop_threshold, bool no_entropy, hipStream_t stream, int32_t device_id)
    : h_sub_mat_(h_sub_mat)
    , sub_mat_dim_(sub_mat_dim)
    , xdrop_threshold_(xdrop_threshold)
    , no_entropy_(no_entropy)
    , stream_(stream)
    , device_id_(device_id)
{
    //TODO - Check bounds
    // Calculate the max limits on the number of extensions we can do on
    // this GPU
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id_);
    const int32_t max_ungapped_per_gb   = 4194304; // FIXME: Calculate using sizeof datastructures
    const int32_t max_seed_pairs_per_gb = 8388608; // FIXME: Calculate using sizeof datastructures
    const float global_mem_gb           = static_cast<float>(device_prop.totalGlobalMem) / 1073741824.0f;
    batch_max_ungapped_extensions_      = static_cast<int32_t>(global_mem_gb) * max_ungapped_per_gb;
    const int32_t max_seed_pairs        = static_cast<int32_t>(global_mem_gb) * max_seed_pairs_per_gb;
    // Switch to device for copying over initial structures
    scoped_device_switch dev(device_id_);

    //Figure out memory requirements for cub functions
    size_t temp_storage_bytes = 0;
    cub_storage_bytes_        = 0;
    GW_CU_CHECK_ERR(hipcub::DeviceSelect::Unique(nullptr, temp_storage_bytes, d_tmp_ssp_, d_tmp_ssp_, (int32_t*)nullptr, batch_max_ungapped_extensions_));
    GW_CU_CHECK_ERR(hipcub::DeviceScan::InclusiveSum(nullptr, cub_storage_bytes_, d_done_, d_done_ + batch_max_ungapped_extensions_, batch_max_ungapped_extensions_));
    cub_storage_bytes_ = std::max(temp_storage_bytes, cub_storage_bytes_);

    // Allocate space on device for scoring matrix and intermediate results
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_sub_mat_, sub_mat_dim_ * sizeof(int32_t)));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_done_, batch_max_ungapped_extensions_ * sizeof(int32_t)));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_tmp_ssp_, batch_max_ungapped_extensions_ * sizeof(ScoredSegmentPair)));
    // Allocate temporary storage for cub
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_temp_storage_cub_, cub_storage_bytes_));

    // Requires pinned host memory registration for proper async behavior
    device_copy_n(h_sub_mat_, sub_mat_dim_, d_sub_mat_, stream_);
    GW_CU_CHECK_ERR(hipMemsetAsync((void*)d_done_, 0, batch_max_ungapped_extensions_ * sizeof(int32_t), stream_));
    GW_CU_CHECK_ERR(hipMemsetAsync((void*)d_tmp_ssp_, 0, batch_max_ungapped_extensions_ * sizeof(ScoredSegmentPair), stream_));
}

StatusType UngappedXDrop::extend_async(const char* d_query, int32_t query_length,
                                       const char* d_target, int32_t target_length,
                                       int32_t score_threshold, SeedPair* d_seed_pairs,
                                       int32_t num_seed_pairs, ScoredSegmentPair* d_scored_segment_pairs,
                                       int32_t* d_num_scored_segment_pairs)
{
    //TODO - Check bounds
    // Switch to configured GPU
    scoped_device_switch dev(device_id_);
    total_scored_segment_pairs_      = 0;
    for (int32_t seed_pair_start = 0; seed_pair_start < num_seed_pairs; seed_pair_start += batch_max_ungapped_extensions_)
    {
        const int32_t curr_num_pairs = std::min(batch_max_ungapped_extensions_, num_seed_pairs - seed_pair_start);
        // TODO- Extricate the kernel launch params?
        find_high_scoring_segment_pairs<<<1024, 128, 0, stream_>>>(d_target,
                                                                   target_length,
                                                                   d_query,
                                                                   query_length,
                                                                   d_sub_mat_,
                                                                   no_entropy_,
                                                                   xdrop_threshold_,
                                                                   score_threshold,
                                                                   d_seed_pairs,
                                                                   curr_num_pairs,
                                                                   seed_pair_start,
                                                                   d_scored_segment_pairs,
                                                                   d_done_);
        GW_CU_CHECK_ERR(hipcub::DeviceScan::InclusiveSum(d_temp_storage_cub_, cub_storage_bytes_, d_done_, d_done_, curr_num_pairs, stream_));
        // TODO- Make async
        const int32_t num_scored_segment_pairs = get_value_from_device(d_done_ + curr_num_pairs - 1, stream_);
        if (num_scored_segment_pairs > 0)
        {
            compress_output<<<1024, 1024, 0, stream_>>>(d_done_,
                                                        seed_pair_start,
                                                        d_scored_segment_pairs,
                                                        d_tmp_ssp_,
                                                        curr_num_pairs); // TODO- Need configurability for kernel?
            thrust::device_ptr<ScoredSegmentPair> d_tmp_hsp_dev_ptr(d_tmp_ssp_);
            // TODO- Make thrust use caching allocator or change kernel
            thrust::stable_sort(thrust::cuda::par.on(stream_),
                                d_tmp_hsp_dev_ptr,
                                d_tmp_hsp_dev_ptr + num_scored_segment_pairs,
                                scored_segment_pair_comp());
            GW_CU_CHECK_ERR(hipcub::DeviceSelect::Unique(d_temp_storage_cub_,
                                                      cub_storage_bytes_,
                                                      d_tmp_ssp_,
                                                      d_scored_segment_pairs + total_scored_segment_pairs_,
                                                      d_num_scored_segment_pairs,
                                                      num_scored_segment_pairs,
                                                      stream_));
            total_scored_segment_pairs_ += get_value_from_device(d_num_scored_segment_pairs, stream_);
        }
    }
    set_device_value_async(d_num_scored_segment_pairs, &total_scored_segment_pairs_, stream_);
    return success;
}

StatusType UngappedXDrop::extend_async(const char* h_query, int32_t query_length,
                                       const char* h_target, int32_t target_length,
                                       int32_t score_threshold,
                                       std::vector<SeedPair>& h_seed_pairs)
{
    // Allocate space on device for target and query sequences, seed_pairs,
    // high scoring segment pairs (ssp) and num_ssp.
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_query_, sizeof(char) * query_length));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_target_, sizeof(char) * target_length));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_seed_pairs_, sizeof(SeedPair) * h_seed_pairs.size()));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_num_ssp_, sizeof(int32_t)));
    GW_CU_CHECK_ERR(hipMalloc((void**)&d_ssp_, sizeof(ScoredSegmentPair) * h_seed_pairs.size()));

    // Async memcopy all the input values to device
    device_copy_n(h_query, query_length, d_query_, stream_);
    device_copy_n(h_target, target_length, d_target_, stream_);
    device_copy_n(h_seed_pairs.data(), h_seed_pairs.size(), d_seed_pairs_, stream_);

    // Launch the ungapped extender device function
    if (!extend_async(d_query_, query_length, d_target_, target_length, score_threshold, d_seed_pairs_, h_seed_pairs.size(), d_ssp_, d_num_ssp_))
    {
        GW_LOG_ERROR("Error running cudaextender");
    }

    return success;
}

StatusType UngappedXDrop::sync()
{
    h_num_ssp_ = get_value_from_device(d_num_ssp_, stream_);
    if (h_num_ssp_ > 0)
    {
        h_ssp_.resize(h_num_ssp_);
        device_copy_n(d_ssp_, h_num_ssp_, &h_ssp_[0], stream_);
        hipStreamSynchronize(stream_);
    }

    return success;
}

const std::vector<ScoredSegmentPair>& UngappedXDrop::get_scored_segment_pairs() const
{
    return h_ssp_;
}

void UngappedXDrop::reset()
{
    // TODO - Add flag for host ptr mode
    // TODO - Add checks for prev free
    h_ssp_.clear();
    GW_CU_CHECK_ERR(hipFree(d_query_));
    GW_CU_CHECK_ERR(hipFree(d_target_));
    GW_CU_CHECK_ERR(hipFree(d_seed_pairs_));
    GW_CU_CHECK_ERR(hipFree(d_num_ssp_));
    GW_CU_CHECK_ERR(hipFree(d_ssp_));
};

UngappedXDrop::~UngappedXDrop()
{
    // TODO - Check flag for host pointer mode
    reset();
    GW_CU_CHECK_ERR(hipFree(d_sub_mat_));
    GW_CU_CHECK_ERR(hipFree(d_tmp_ssp_));
    GW_CU_CHECK_ERR(hipFree(d_done_));
};

} // namespace cudaextender

} // namespace genomeworks

} // namespace claraparabricks