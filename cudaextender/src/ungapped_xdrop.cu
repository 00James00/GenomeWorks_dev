#include "hip/hip_runtime.h"
/*
* Copyright 2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include "ungapped_xdrop.cuh"
#include "ungapped_xdrop_kernels.cuh"

#include <claraparabricks/genomeworks/utils/pinned_host_vector.hpp>
#include <claraparabricks/genomeworks/utils/cudautils.hpp>

#include <thrust/system/cuda/execution_policy.h>
#include <thrust/sort.h>

#include <cub/device/device_select.cuh>
#include <cub/device/device_scan.cuh>

namespace claraparabricks
{

namespace genomeworks
{

namespace cudaextender
{

using namespace cudautils;

UngappedXDrop::UngappedXDrop(const int32_t* h_sub_mat, const int32_t sub_mat_dim, const int32_t xdrop_threshold, const bool no_entropy, hipStream_t stream, const int32_t device_id, DefaultDeviceAllocator allocator)
    : h_sub_mat_(h_sub_mat)
    , sub_mat_dim_(sub_mat_dim)
    , xdrop_threshold_(xdrop_threshold)
    , no_entropy_(no_entropy)
    , stream_(stream)
    , device_id_(device_id)
    , host_ptr_api_mode_(false)
    , allocator_(allocator)
{
    if (h_sub_mat_ == nullptr)
    {
        throw std::runtime_error("Substitution matrix cannot be null");
    }
    // TODO - check sub_mat_dim based on Sequence Encoder API
    // Calculate the max limits on the number of extensions we can do on this GPU
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id_);
    const int32_t max_ungapped_per_gb = 4194304; // FIXME: Calculate using sizeof datastructures
    //const int32_t max_seed_pairs_per_gb = 8388608; // FIXME: Calculate using sizeof datastructures // TODO- Do we need this?
    const float global_mem_gb      = static_cast<float>(device_prop.totalGlobalMem) / 1073741824.0f;
    batch_max_ungapped_extensions_ = static_cast<int32_t>(global_mem_gb) * max_ungapped_per_gb;
    // Switch to device for copying over initial structures
    scoped_device_switch dev(device_id_);

    //Figure out memory requirements for cub functions
    size_t temp_storage_bytes = 0;
    size_t cub_storage_bytes  = 0;
    GW_CU_CHECK_ERR(hipcub::DeviceSelect::Unique(nullptr, temp_storage_bytes, d_tmp_ssp_.data(), d_tmp_ssp_.data(), (int32_t*)nullptr, batch_max_ungapped_extensions_, stream_));
    GW_CU_CHECK_ERR(hipcub::DeviceScan::InclusiveSum(nullptr, cub_storage_bytes, d_done_.data(), d_done_.data(), batch_max_ungapped_extensions_, stream_));
    cub_storage_bytes = std::max(temp_storage_bytes, cub_storage_bytes);

    // Allocate space on device for scoring matrix and intermediate results
    d_sub_mat_          = device_buffer<int32_t>(sub_mat_dim_, allocator_, stream_);
    d_done_             = device_buffer<int32_t>(batch_max_ungapped_extensions_, allocator_, stream_);
    d_tmp_ssp_          = device_buffer<ScoredSegmentPair>(batch_max_ungapped_extensions_, allocator_, stream_);
    d_temp_storage_cub_ = device_buffer<char>(cub_storage_bytes, allocator_, stream_);

    // Requires pinned host memory registration for proper async behavior
    device_copy_n(h_sub_mat_, sub_mat_dim_, d_sub_mat_.data(), stream_);
}

StatusType UngappedXDrop::extend_async(const char* d_query, int32_t query_length,
                                       const char* d_target, int32_t target_length,
                                       int32_t score_threshold, SeedPair* d_seed_pairs,
                                       int32_t num_seed_pairs, ScoredSegmentPair* d_scored_segment_pairs,
                                       int32_t* d_num_scored_segment_pairs)
{
    if (d_query == nullptr || d_target == nullptr || d_seed_pairs == nullptr)
    {
        GW_LOG_ERROR("Invalid input pointers");
        return StatusType::invalid_input;
    }
    if (d_scored_segment_pairs == nullptr || d_num_scored_segment_pairs == nullptr)
    {
        GW_LOG_ERROR("Invalid output pointers");
        return StatusType::invalid_input;
    }
    // Switch to configured GPU
    scoped_device_switch dev(device_id_);
    total_scored_segment_pairs_ = 0;
    for (int32_t seed_pair_start = 0; seed_pair_start < num_seed_pairs; seed_pair_start += batch_max_ungapped_extensions_)
    {
        // TODO - Do we need these? It seems we don't!
        GW_CU_CHECK_ERR(hipMemsetAsync((void*)d_done_.data(), 0, batch_max_ungapped_extensions_ * sizeof(int32_t), stream_));
        GW_CU_CHECK_ERR(hipMemsetAsync((void*)d_tmp_ssp_.data(), 0, batch_max_ungapped_extensions_ * sizeof(ScoredSegmentPair), stream_));
        const int32_t curr_num_pairs = std::min(batch_max_ungapped_extensions_, num_seed_pairs - seed_pair_start);
        // TODO- Extricate the kernel launch params?
        find_high_scoring_segment_pairs<<<1024, 128, 0, stream_>>>(d_target,
                                                                   target_length,
                                                                   d_query,
                                                                   query_length,
                                                                   d_sub_mat_.data(),
                                                                   no_entropy_,
                                                                   xdrop_threshold_,
                                                                   score_threshold,
                                                                   d_seed_pairs,
                                                                   curr_num_pairs,
                                                                   seed_pair_start,
                                                                   d_scored_segment_pairs,
                                                                   d_done_.data());
        size_t cub_storage_bytes = d_temp_storage_cub_.size();
        GW_CU_CHECK_ERR(hipcub::DeviceScan::InclusiveSum(d_temp_storage_cub_.data(), cub_storage_bytes, d_done_.data(), d_done_.data(), curr_num_pairs, stream_))
        // TODO- Make async
        const int32_t num_scored_segment_pairs = get_value_from_device(d_done_.data() + curr_num_pairs - 1, stream_);
        if (num_scored_segment_pairs > 0)
        {
            compress_output<<<1024, 1024, 0, stream_>>>(d_done_.data(),
                                                        seed_pair_start,
                                                        d_scored_segment_pairs,
                                                        d_tmp_ssp_.data(),
                                                        curr_num_pairs); // TODO- Need configurability for kernel?
            thrust::stable_sort(thrust::cuda::par(allocator_).on(stream_),
                                d_tmp_ssp_.begin(),
                                d_tmp_ssp_.begin() + num_scored_segment_pairs,
                                scored_segment_pair_comp());
            GW_CU_CHECK_ERR(hipcub::DeviceSelect::Unique(d_temp_storage_cub_.data(),
                                                      cub_storage_bytes,
                                                      d_tmp_ssp_.data(),
                                                      d_scored_segment_pairs + total_scored_segment_pairs_,
                                                      d_num_scored_segment_pairs,
                                                      num_scored_segment_pairs,
                                                      stream_))
            total_scored_segment_pairs_ += get_value_from_device(d_num_scored_segment_pairs, stream_);
        }
    }

    set_device_value_async(d_num_scored_segment_pairs, &total_scored_segment_pairs_, stream_);

    return StatusType::success;
}

StatusType UngappedXDrop::extend_async(const char* h_query, const int32_t& query_length,
                                       const char* h_target, const int32_t& target_length,
                                       const int32_t& score_threshold,
                                       const std::vector<SeedPair>& h_seed_pairs)
{
    // Reset the extender if it was used before in this mode
    reset();
    // Set host pointer mode on
    host_ptr_api_mode_ = true;
    // Allocate space for query and target sequences
    d_query_  = device_buffer<char>(query_length, allocator_, stream_);
    d_target_ = device_buffer<char>(target_length, allocator_, stream_);
    // Allocate space for SeedPair input
    d_seed_pairs_ = device_buffer<SeedPair>(h_seed_pairs.size(), allocator_, stream_);
    // Allocate space for ScoredSegmentPair output
    d_ssp_     = device_buffer<ScoredSegmentPair>(h_seed_pairs.size(), allocator_, stream_);
    d_num_ssp_ = device_buffer<int32_t>(1, allocator_, stream_);

    // Async memcopy all the input values to device
    device_copy_n(h_query, query_length, d_query_.data(), stream_);
    device_copy_n(h_target, target_length, d_target_.data(), stream_);
    device_copy_n(h_seed_pairs.data(), h_seed_pairs.size(), d_seed_pairs_.data(), stream_);

    // Launch the ungapped extender device function
    return extend_async(d_query_.data(), query_length,
                        d_target_.data(), target_length,
                        score_threshold, d_seed_pairs_.data(),
                        d_seed_pairs_.size(), d_ssp_.data(),
                        d_num_ssp_.data());
}

StatusType UngappedXDrop::sync()
{
    if (host_ptr_api_mode_)
    {
        const int32_t h_num_ssp = get_value_from_device(d_num_ssp_.data(), stream_);
        if (h_num_ssp > 0)
        {
            h_ssp_.resize(h_num_ssp);
            device_copy_n(d_ssp_.data(), h_num_ssp, h_ssp_.data(), stream_);
            hipStreamSynchronize(stream_);
        }
        return StatusType::success;
    }

    // If this function was called without using the host_ptr_api, throw error
    return StatusType::invalid_operation;
}

const std::vector<ScoredSegmentPair>& UngappedXDrop::get_scored_segment_pairs() const
{
    if (host_ptr_api_mode_)
    {
        return h_ssp_;
    }
    // If this function was called using the host_ptr_api, throw error
    throw std::runtime_error("Invalid API call. Getting scored segment pairs without calling extend_async host ptr API");
}

void UngappedXDrop::reset()
{
    // Reset these only if host pointer API was used earlier
    if (host_ptr_api_mode_)
    {
        h_ssp_.clear();
        host_ptr_api_mode_ = false;
    }
}

UngappedXDrop::~UngappedXDrop()
{
    UngappedXDrop::reset();
}

} // namespace cudaextender

} // namespace genomeworks

} // namespace claraparabricks
